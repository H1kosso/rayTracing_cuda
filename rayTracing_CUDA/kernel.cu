#include "hip/hip_runtime.h"
#include "Kernel.h"

#define gpuErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
static inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
	if (code != hipSuccess) {
		fprintf(stderr, "GPU assert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) {
			std::exit(code);
		}
	}
}

static __global__ void sum(uint32_t* d_a, const uint32_t N) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	if (index < N) {
		d_a[index] = index;
		d_a[index] |= 0xff000000;
	}
}

Kernel::Kernel(): kernelTimeMs(0.f), TPB(64){
}

void Kernel::runKernel() {
	// TODO: Je�li to b�dzie w p�tli si� od�wie�a�o to warto nie alokowa� tego za ka�dym razem
	uint32_t* d_buffer = nullptr;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	if (!bufferSize){
		throw std::invalid_argument("buffer_size is NULL!");
	}
	else if (!buffer) {
		throw std::invalid_argument("buffer is NULL!");
	}

	gpuErrChk(hipMalloc(&d_buffer,  bufferSize * sizeof(*d_buffer)));

	gpuErrChk(hipMemcpy(d_buffer, buffer, bufferSize * sizeof(*d_buffer), hipMemcpyHostToDevice));

	hipEventRecord(start);
	sum <<< (bufferSize + TPB - 1) / TPB, TPB >>> (d_buffer, bufferSize);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&kernelTimeMs, start, stop);

	gpuErrChk(hipMemcpy(buffer, d_buffer, bufferSize * sizeof(*d_buffer), hipMemcpyDeviceToHost));

	gpuErrChk(hipFree(d_buffer));
}

float Kernel::getKernelTimeMs()
{
	return kernelTimeMs;
}

Kernel::~Kernel() {}

void Kernel::setBufferSize(uint32_t size){
	this->bufferSize = size;
}

void Kernel::setBuffer(uint32_t* buffer)
{
	this->buffer = buffer;
}
