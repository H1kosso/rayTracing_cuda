#include "hip/hip_runtime.h"
#include "Kernel.h"

#define gpuErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
static inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
	if (code != hipSuccess) {
		fprintf(stderr, "GPU assert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) {
			hipDeviceReset(); // destroys all allocations, resets all states
			std::exit(code);
		}
	}
}

static __global__ void sum(uint32_t* d_a, const glm::uvec2 imgDim) {
	uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	uint32_t index = x + y * blockDim.x * gridDim.x;

	if (imgDim.x <= x || imgDim.y <= y || imgDim.x * imgDim.y <= index) {
		return;
	}
	glm::vec2 coord = { (float)x / (float)imgDim.x, (float)y / (float)imgDim.y}; // [0; 1]
	coord *= 2.f - 1.f; // [-1; 1]
	uint8_t r = (uint8_t)(coord.x * 255.0f);
	uint8_t g = (uint8_t)(coord.y * 255.0f);

	d_a[index] = 0xff000000 | (g << 8) | r;
}

Kernel::Kernel(): kernelTimeMs(0.f), TPB(16){
}

void Kernel::runKernel() {
	// TODO: Je�li to b�dzie w p�tli si� od�wie�a�o to warto nie alokowa� tego za ka�dym razem
	uint32_t* d_buffer = nullptr;
	uint32_t bufferSize = imgDim.x * imgDim.y;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	if (!bufferSize){
		throw std::invalid_argument("CUDA: buffer size is not set!");
	}
	else if (!buffer) {
		throw std::invalid_argument("CUDA: buffer is NULL!");
	}

	gpuErrChk(hipMalloc(&d_buffer,  bufferSize * sizeof(*d_buffer)));
	gpuErrChk(hipMemcpy(d_buffer, buffer, bufferSize * sizeof(*d_buffer), hipMemcpyHostToDevice));

	dim3 gridDim((imgDim.x + TPB - 1) / TPB, (imgDim.y + TPB - 1) / TPB);
	dim3 blockDim(TPB, TPB);

	hipEventRecord(start);
	sum << < gridDim, blockDim >> > (d_buffer, imgDim);
	gpuErrChk(hipGetLastError());
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&kernelTimeMs, start, stop);

	gpuErrChk(hipMemcpy(buffer, d_buffer, bufferSize * sizeof(*d_buffer), hipMemcpyDeviceToHost));

	gpuErrChk(hipFree(d_buffer));
}

float Kernel::getKernelTimeMs()
{
	return kernelTimeMs;
}

Kernel::~Kernel() {}

void Kernel::setImgDim(glm::uvec2 imgDim){
	this->imgDim = imgDim;
}

void Kernel::setBuffer(uint32_t* buffer)
{
	this->buffer = buffer;
}
