#include "hip/hip_runtime.h"
#include "Kernel.h"

static __global__ void trace_ray(uint32_t* d_imgBuff, const glm::uvec2 imgDim, const Sphere* d_hittable, const uint32_t hittableSize) {
	uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	uint32_t gIndex = x + y * blockDim.x * gridDim.x;

	if (imgDim.x <= x || imgDim.y <= y || imgDim.x * imgDim.y <= gIndex) {
		return;
	}
	glm::vec2 coord = { ((float)x * 2.f / (float)imgDim.x) - 1.f, ((float)y * 2.f / (float)imgDim.y) - 1.f}; // [-1; 1]

	if (!hittableSize) {
		d_imgBuff[gIndex] = convertFromRGBA({0.f, 0.f, 0.f, 1.f});
		return;
	}

	const glm::vec3 rayOrigin = { 0.f, 0.f, 1.f };
	glm::vec3 rayDirection = glm::normalize(glm::vec3(coord.x, coord.y, -1.f));

	const Sphere* closestSphere = nullptr;
	glm::vec3 closestShiftOrigin{};
	float closestT{FLT_MAX};

	for (int i = 0; i < hittableSize; i++) {
		// Shifing current camera to the position of given object. It's for the calculation of intersections.
		glm::vec3 shiftOrigin = rayOrigin - d_hittable[i].getPosition();
		float t = d_hittable[i].hit(shiftOrigin, rayDirection);
		if (t < 0.f)
			continue;

		if (t < closestT) {
			closestSphere = &d_hittable[i];
			closestT = t;
			closestShiftOrigin = shiftOrigin;
		}
	}

	if (closestSphere == nullptr) {
		d_imgBuff[gIndex] = convertFromRGBA({0.f, 0.f, 0.f, 1.f});
		return;
	}

	glm::vec3 closestHit = closestT * rayDirection + closestShiftOrigin;
	glm::vec3 normal = glm::normalize(closestHit); // normal as unit vector of closestHit

	glm::vec3 lightSource = glm::normalize(glm::vec3(1.f, 1.f, -1.f));
	float lightIntensity = glm::max(glm::dot(normal, -lightSource), 0.f); // only angles: 0 <= d <= 90

	d_imgBuff[gIndex] = convertFromRGBA(
		{
			closestSphere->getColor().r * lightIntensity,
			closestSphere->getColor().g * lightIntensity,
			closestSphere->getColor().b * lightIntensity,
			closestSphere->getColor().a 
		});
	// d_imgBuff[gIndex] = convertFromRGBA(closestSphere->getColor() * lightIntensity);
}


Kernel::Kernel(): kernelTimeMs(0.f), TPB(16){
}

void Kernel::runKernel(Scene& scene) {
	// TODO: Je�li to b�dzie w p�tli si� od�wie�a�o to warto nie alokowa� tego za ka�dym razem
	uint32_t* d_buffer = nullptr;
	Sphere* d_hittable = nullptr;
	uint32_t bufferSize = imgDim.x * imgDim.y;
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	if (!bufferSize){
		throw std::invalid_argument("CUDA: buffer size is not set!");
	}
	else if (!buffer) {
		throw std::invalid_argument("CUDA: buffer is NULL!");
	}

	gpuErrChk(hipMalloc(&d_buffer,  bufferSize * sizeof(*d_buffer)));
	gpuErrChk(hipMalloc(&d_hittable, scene.sphere.size() * sizeof(*d_hittable)));

	gpuErrChk(hipMemcpy(d_buffer, buffer, bufferSize * sizeof(*d_buffer), hipMemcpyHostToDevice));
	gpuErrChk(hipMemcpy(d_hittable, scene.sphere.data(), scene.sphere.size() * sizeof(*d_hittable), hipMemcpyHostToDevice));

	dim3 gridDim((imgDim.x + TPB - 1) / TPB, (imgDim.y + TPB - 1) / TPB);
	dim3 blockDim(TPB, TPB);

	hipEventRecord(start);
	trace_ray << < gridDim, blockDim >> > (d_buffer, imgDim, d_hittable, scene.sphere.size());
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&kernelTimeMs, start, stop);
	gpuErrChk(hipGetLastError());

	gpuErrChk(hipMemcpy(buffer, d_buffer, bufferSize * sizeof(*d_buffer), hipMemcpyDeviceToHost));

	gpuErrChk(hipFree(d_buffer));
	gpuErrChk(hipFree(d_hittable));
}

float Kernel::getKernelTimeMs()
{
	return kernelTimeMs;
}

Kernel::~Kernel() {}

void Kernel::setImgDim(glm::uvec2 imgDim){
	this->imgDim = imgDim;
}

void Kernel::setBuffer(uint32_t* buffer)
{
	this->buffer = buffer;
}
