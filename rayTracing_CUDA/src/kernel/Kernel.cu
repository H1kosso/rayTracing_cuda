#include "hip/hip_runtime.h"
#include "Kernel.h"

static __global__ void trace_ray(uint32_t *d_imgBuff, const glm::uvec2 imgDim,
                                 const Sphere *d_hittable,
                                 const uint32_t hittableSize,
                                 const glm::vec3 cameraOrigin,
                                 glm::vec3 *d_rayDirections,
                                 int size_rayDirections) {
  uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
  uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
  uint32_t gIndex = x + y * blockDim.x * gridDim.x;

  if (imgDim.x <= x || imgDim.y <= y || imgDim.x * imgDim.y <= gIndex) {
    return;
  }
  glm::vec2 coord = {((float)x * 2.f / (float)imgDim.x) - 1.f,
                     ((float)y * 2.f / (float)imgDim.y) - 1.f}; // [-1; 1]

  float grad = 0.5f * (-coord.y + 1.f);
  glm::vec4 backgroundColor = {(1.f - grad) * glm::vec3(1.f, 1.f, 1.f) +
                                   grad * glm::vec3(0.5f, 0.7f, 1.0f),
                               1.f};

  if (!hittableSize) {
    d_imgBuff[gIndex] = convertFromRGBA(backgroundColor);
    return;
  }

  Ray ray;
  ray.origin = cameraOrigin;
  ray.direction = glm::normalize(d_rayDirections[x + y * imgDim.x]);


  if (!hittableSize) {
    d_imgBuff[gIndex] = convertFromRGBA({0.f, 0.f, 0.f, 1.f});
    return;
  }

  const Sphere *closestSphere = nullptr;
  glm::vec3 closestShiftOrigin{};
  float closestT{FLT_MAX};


    for (int i = 0; i < hittableSize; i++) {
      // Shifing current camera to the position of given object. It's used for
      // the calculation of intersections.
      glm::vec3 shiftOrigin = ray.origin - d_hittable[i].getPosition();
      float t = d_hittable[i].hit({shiftOrigin, ray.direction});
      if (t < 0.f)
        continue;
      if (t < closestT) {
        closestSphere = &d_hittable[i];
        closestT = t;
        closestShiftOrigin = shiftOrigin;
      }
    }

    if (closestSphere == nullptr) {
      d_imgBuff[gIndex] = convertFromRGBA(backgroundColor);
      return;
    }

    glm::vec3 closestHit = closestT * ray.direction + closestShiftOrigin;
    glm::vec3 normal = glm::normalize(closestHit); // normal as unit vector of closestHit
    

    glm::vec3 lightSource = glm::normalize(glm::vec3(1.f, 1.f, -1.f));
    float lightIntensity = glm::max(glm::dot(normal, -lightSource),
                                    0.f); // only angles: 0 <= d <= 90

    d_imgBuff[gIndex] =
        convertFromRGBA({closestSphere->getColor().r * lightIntensity,
                         closestSphere->getColor().g * lightIntensity,
                         closestSphere->getColor().b * lightIntensity,
                         closestSphere->getColor().a});
    // d_imgBuff[gIndex] = convertFromRGBA(closestSphere->getColor() *
    // lightIntensity);
  }

  Kernel::Kernel() : kernelTimeMs(0.f), TPB(16) {}

  void Kernel::runKernel(Scene & scene, Camera camera) {
    // TODO: Je�li to b�dzie w p�tli si� od�wie�a�o to warto nie alokowa� tego
    // za ka�dym razem
    uint32_t *d_buffer = nullptr;
    Sphere *d_hittable = nullptr;
    glm::vec3 *d_vec3 = nullptr;
    uint32_t bufferSize = imgDim.x * imgDim.y;
    hipEvent_t start, stop;

    std::vector<glm::vec3> rayDirections = camera.GetRayDirections();

    hipEventCreate(&start);
    hipEventCreate(&stop);
    if (!bufferSize) {
      throw std::invalid_argument("CUDA: buffer size is not set!");
    } else if (!buffer) {
      throw std::invalid_argument("CUDA: buffer is NULL!");
    }

    gpuErrChk(hipMalloc(&d_buffer, bufferSize * sizeof(*d_buffer)));
    gpuErrChk(
        hipMalloc(&d_hittable, scene.sphere.size() * sizeof(*d_hittable)));
    gpuErrChk(hipMalloc(&d_vec3, rayDirections.size() * sizeof(glm::vec3)));

    gpuErrChk(hipMemcpy(d_buffer, buffer, bufferSize * sizeof(*d_buffer),
                         hipMemcpyHostToDevice));
    gpuErrChk(hipMemcpy(d_hittable, scene.sphere.data(),
                         scene.sphere.size() * sizeof(*d_hittable),
                         hipMemcpyHostToDevice));
    gpuErrChk(hipMemcpyAsync(d_vec3, rayDirections.data(),
                              rayDirections.size() * sizeof(glm::vec3),
                              hipMemcpyHostToDevice))

        dim3 gridDim((imgDim.x + TPB - 1) / TPB, (imgDim.y + TPB - 1) / TPB);
    dim3 blockDim(TPB, TPB);

    hipEventRecord(start);
    trace_ray<<<gridDim, blockDim>>>(d_buffer, imgDim, d_hittable,
                                     scene.sphere.size(), camera.GetPosition(),
                                     d_vec3, rayDirections.size());

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&kernelTimeMs, start, stop);
    gpuErrChk(hipGetLastError());

    gpuErrChk(hipMemcpy(buffer, d_buffer, bufferSize * sizeof(*d_buffer),
                         hipMemcpyDeviceToHost));

    gpuErrChk(hipFree(d_buffer));
    gpuErrChk(hipFree(d_hittable));
    gpuErrChk(hipFree(d_vec3));
  }


  float Kernel::getKernelTimeMs() { return kernelTimeMs; }

  Kernel::~Kernel() {}

  void Kernel::setImgDim(glm::uvec2 imgDim) { this->imgDim = imgDim; }

  void Kernel::setBuffer(uint32_t * buffer) { this->buffer = buffer; }
