#include "hip/hip_runtime.h"
#include "Kernel.h"

static __global__ void init_curand(hiprandStatePhilox4_32_10_t* states, const glm::uvec2 imgDim) {
	uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	uint32_t gIndex = x + y * blockDim.x * gridDim.x;

	if (imgDim.x <= x || imgDim.y <= y || imgDim.x * imgDim.y <= gIndex) {
		return;
	}
	hiprand_init((size_t)gIndex, 0, 0, &states[gIndex]);
	// Sequence 0 and offset 0 for better performance but may result in worse 'randomness'
}

static __global__ void trace_ray(
	uint32_t* imgBuff,
	const glm::uvec2 imgDim,
	hiprandStatePhilox4_32_10_t* rndState,
	const Sphere* hittable,
	const uint32_t hittableSize,
	const glm::vec3 cameraOrigin,
	glm::vec3* d_rayDirections,
	int size_rayDirections) {

	uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	uint32_t gIndex = x + y * blockDim.x * gridDim.x;

  if (imgDim.x <= x || imgDim.y <= y || imgDim.x * imgDim.y <= gIndex) {
    return;
  }
  glm::vec2 coord = {((float)x * 2.f / (float)imgDim.x) - 1.f,
                     ((float)y * 2.f / (float)imgDim.y) - 1.f}; // [-1; 1]

  float grad = 0.5f * (-coord.y + 1.f);
  glm::vec4 backgroundColor = {(1.f - grad) * glm::vec3(1.f, 1.f, 1.f) +
                                   grad * glm::vec3(0.5f, 0.7f, 1.0f),
                               1.f};

	if (!hittableSize) {
		imgBuff[gIndex] = convertFromRGBA(backgroundColor);
		return;
	}

  Ray ray;
  ray.origin = cameraOrigin;
  ray.direction = glm::normalize(d_rayDirections[x + y * imgDim.x]);


  if (!hittableSize) {
    imgBuff[gIndex] = convertFromRGBA({0.f, 0.f, 0.f, 1.f});
    return;
  }

  const Sphere *closestSphere = nullptr;
  glm::vec3 closestShiftOrigin{};
  float closestT{FLT_MAX};

	for (int i = 0; i < hittableSize; i++) {
		// Shifing current camera to the position of given object. It's used for the calculation of intersections.
		glm::vec3 shiftOrigin = ray.origin - hittable[i].getPosition();
		float t = hittable[i].hit({ shiftOrigin, ray.direction });
		if (t < 0.f)
			continue;

		if (t < closestT) {
			closestSphere = &hittable[i];
			closestT = t;
			closestShiftOrigin = shiftOrigin;
		}
	}

	if (closestSphere == nullptr) {
		imgBuff[gIndex] = convertFromRGBA(backgroundColor);
		return;
	}

	glm::vec3 closestHit = closestT * ray.direction + closestShiftOrigin;
	glm::vec3 normal = glm::normalize(closestHit); // normal as unit vector of closestHit

	glm::vec3 lightSource = glm::normalize(glm::vec3(1.f, 1.f, -1.f));
	float lightIntensity = glm::max(glm::dot(normal, -lightSource), 0.f); // only angles: 0 <= d <= 90

	imgBuff[gIndex] = convertFromRGBA(
		{
			closestSphere->getColor().r * lightIntensity,
			closestSphere->getColor().g * lightIntensity,
			closestSphere->getColor().b * lightIntensity,
			closestSphere->getColor().a 
		});
	// d_imgBuff[gIndex] = convertFromRGBA(closestSphere->getColor() * lightIntensity);
}


Kernel::Kernel(): kernelTimeMs(0.f), TPB(16){
}

void Kernel::runKernel(const Scene& scene, const Camera& camera) {
	// TODO: Je�li to b�dzie w p�tli si� od�wie�a�o to warto nie alokowa� tego za ka�dym razem
	uint32_t* d_buffer = nullptr;
	Sphere* d_hittable = nullptr;
	hiprandStatePhilox4_32_10_t* d_curandState = nullptr;
	glm::vec3* d_rayDirections = nullptr;
	uint32_t bufferSize = imgDim.x * imgDim.y;
	hipEvent_t start, stop;
	dim3 gridDim((imgDim.x + TPB - 1) / TPB, (imgDim.y + TPB - 1) / TPB);
	dim3 blockDim(TPB, TPB);

	std::vector<glm::vec3> rayDirections = camera.GetRayDirections();

    hipEventCreate(&start);
    hipEventCreate(&stop);
    if (!bufferSize) {
      throw std::invalid_argument("CUDA: buffer size is not set!");
    } else if (!buffer) {
      throw std::invalid_argument("CUDA: buffer is NULL!");
    }

	gpuErrChk(hipMalloc(&d_buffer,  bufferSize * sizeof(*d_buffer)));
	gpuErrChk(hipMalloc(&d_hittable, scene.sphere.size() * sizeof(*d_hittable)));
	gpuErrChk(hipMalloc(&d_curandState, bufferSize * sizeof(*d_curandState)));
	gpuErrChk(hipMalloc(&d_rayDirections, rayDirections.size() * sizeof(glm::vec3)));

	hipEventRecord(start);

    gpuErrChk(hipMemcpy(d_buffer, buffer, bufferSize * sizeof(*d_buffer),
                         hipMemcpyHostToDevice));
    gpuErrChk(hipMemcpy(d_hittable, scene.sphere.data(),
                         scene.sphere.size() * sizeof(*d_hittable),
                         hipMemcpyHostToDevice));
    gpuErrChk(hipMemcpyAsync(d_rayDirections, rayDirections.data(),
                              rayDirections.size() * sizeof(glm::vec3),
                              hipMemcpyHostToDevice))

	trace_ray << < gridDim, blockDim >> > (
		d_buffer,
		imgDim, d_curandState,
		d_hittable,
		scene.sphere.size(),
		camera.GetPosition(),
		d_rayDirections,
		rayDirections.size());

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&kernelTimeMs, start, stop);
	gpuErrChk(hipGetLastError());

    gpuErrChk(hipMemcpy(buffer, d_buffer, bufferSize * sizeof(*d_buffer),
                         hipMemcpyDeviceToHost));

	gpuErrChk(hipFree(d_buffer));
	gpuErrChk(hipFree(d_hittable));
	gpuErrChk(hipFree(d_curandState));
	gpuErrChk(hipFree(d_rayDirections));
}


  float Kernel::getKernelTimeMs() { return kernelTimeMs; }

  Kernel::~Kernel() {}

  void Kernel::setImgDim(glm::uvec2 imgDim) { this->imgDim = imgDim; }

  void Kernel::setBuffer(uint32_t* buffer) { this->buffer = buffer; }
